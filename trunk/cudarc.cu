#include "hip/hip_runtime.h"
/**
* Fabio Markus Miranda
* fmiranda@tecgraf.puc-rio.br
* fabiom@gmail.com
* Dec 2010
* 
* cudarc.cu: CUDA functions
*/

#define EPSILON 0

#ifndef CUDARC_WINGL
	#include <windows.h>
#else
	#include <GL/glut.h>
#endif

#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
#include <cutil_inline.h>
#include <cutil_math.h>
#include <hip/hip_math_constants.h>

#include "defines.h"
#include "cudamemory.cuh"

enum InterpolType {Const = 0, Linear = 1, Quad = 2, Step = 3};

struct Elem{
  float4 interpolfunc0;
#ifdef CUDARC_HEX
  float4 interpolfunc1;
#endif
};

struct Ray{
  float4 acccolor;
  float4 dir;
  float4 eyepos;
  float t;
  int frontid;
  int frontface;
  float frontscalar;
  Elem currentelem;
};

inline __host__ __device__ float4 operator*(float4 a, float4 b)
{
  return make_float4(a.x * b.x, a.y * b.y, a.z * b.z, a.w * b.w);
}

inline __host__ __device__ float4 cross(float4 a, float4 b)
{ 
  return make_float4(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x, 0); 
}


inline __host__ __device__ float permuted_inner_produtct(float4 pr, float4 qr, float4 ps, float4 qs)
{
  return dot(pr, qs) + dot(qr, ps);
}


#ifdef CUDARC_HEX

/**
* Ray bilinear patch intersection (hexahedral mesh)
*/

__device__ float ComputeU(float v, float a1, float a2, float b1, float b2, float c1, float c2, float d1, float d2){
  float a = v * a2 + b2;
  float b = v * (a2 - a1) + b2 - b1;

  if(fabs(b) > fabs(a))
    return (v * (c1 - c2) + d1 - d2) / b;
  else
    return (- v * c2 - d2) / a;
}

__device__ float ComputeT(Ray* ray, float4 p){
  /*
  if(fabs(ray->dir.x) >= fabs(ray->dir.y) && fabs(ray->dir.x) >= fabs(ray->dir.z))
  return (p.x - ray->eyepos.x) / ray->dir.x;
  else if(fabs(ray->dir.y) >= fabs(ray->dir.z))
  return (p.y - ray->eyepos.y) / ray->dir.y;
  else
  return (p.z - ray->eyepos.z) / ray->dir.z;
  */
  //p.w = 1;
  //ray->eyepos.w = 1;
  return length(p - ray->eyepos);
} 

__device__ float Solve(Ray* ray, float4 v00, float4 v01, float4 v10, float4 v11, float v, float a1, float a2, float b1, float b2, float c1, float c2, float d1, float d2){

  if(v >= -EPSILON && v <= 1.0f + EPSILON){
    float u = ComputeU(v, a1, a2, b1, b2, c1, c2, d1, d2);
    if(u >= -EPSILON && u <= 1.0f + EPSILON){
      float4 p = (1.0f - u) * (1.0f - v) * v00 + v * (1.0f - u) * v01 + u * (1.0f - v) * v10 + u * v * v11;
      float t = ComputeT(ray, p);
      //if(t >= -EPSILON)
        return t;
      //return length(p - ray->eyepos);
    }
  }
  //return u;
  return HIP_INF_F;

}

__device__ float2 Intersect(Ray* ray, float4 v00, float4 v01, float4 v10, float4 v11, bool getfart){


  float4 a = v11 - v10 - v01 + v00;
  float4 b = v10 - v00;
  float4 c = v01 - v00;
  float4 d = v00 - ray->eyepos;

  float a1 = a.x * ray->dir.z - a.z * ray->dir.x;
  float b1 = b.x * ray->dir.z - b.z * ray->dir.x;
  float c1 = c.x * ray->dir.z - c.z * ray->dir.x;
  float d1 = d.x * ray->dir.z - d.z * ray->dir.x;
  float a2 = a.y * ray->dir.z - a.z * ray->dir.y;
  float b2 = b.y * ray->dir.z - b.z * ray->dir.y;
  float c2 = c.y * ray->dir.z - c.z * ray->dir.y;
  float d2 = d.y * ray->dir.z - d.z * ray->dir.y;

  //Solve the equation (A2C1 - A1C2) * v^2 + (A2D1 - A1D2 + B2C1 - B1C2) * v + (B2D1 - B1D2) = 0
  float aux_a = a2 * c1 - a1 * c2;
  float aux_b = a2 * d1 - a1 * d2 + b2 * c1 - b1 * c2;
  float aux_c = b2 * d1 - b1 * d2;
  float delta = aux_b * aux_b - 4.0f * (aux_a) * (aux_c);

#if 0
  //a close to zero
  //if(aux_a == 0.0f){
  if(aux_a >= -EPSILON && aux_a <= EPSILON){
    //if(aux_b != 0.0f){
    if(aux_b <= -EPSILON || aux_b >= EPSILON){
      float root = -aux_c / aux_b;
      if(root > -EPSILON && root < 1 + EPSILON){
        //return 1;
        //return Solve(ray, v00, v01, v10, v11, root, a1, a2, b1, b2, c1, c2, d1, d2);
        return make_float2(Solve(ray, v00, v01, v10, v11, root, a1, a2, b1, b2, c1, c2, d1, d2), 1); //cai aqui
      }
      else{
        //return 0;
        //return HIP_INF_F;
        return make_float2(HIP_INF_F, 0);
      }
    }
    else{
      //return 0;
      //return HIP_INF_F;
      return make_float2(HIP_INF_F, 0);
    }
  }

  if(delta <= EPSILON){
    if(delta >= -EPSILON && delta <= EPSILON){
      float root = -aux_b / aux_a;
      if(root > -EPSILON && root < 1 + EPSILON){
        //return 1;
        //return Solve(ray, v00, v01, v10, v11, root, a1, a2, b1, b2, c1, c2, d1, d2);
        return make_float2(Solve(ray, v00, v01, v10, v11, root, a1, a2, b1, b2, c1, c2, d1, d2), 1);
      }
      else{
        //return 0;
        //return HIP_INF_F;
        return make_float2(HIP_INF_F, 0);
      }
    }
    else{
      //return 0;
      //return HIP_INF_F;
      return make_float2(HIP_INF_F, 0);
    }
  }

  float q;
  if(aux_b < EPSILON)
    q = - 0.5f * (aux_b - sqrtf(delta));
  else
    q = - 0.5f * (aux_b + sqrtf(delta));

  float root1 = q / aux_a;
  float root2 = aux_c / q;

  if(root1 > -EPSILON && root1 < 1 + EPSILON && root2 > -EPSILON && root2 < 1 + EPSILON){
    //return 2;
    float t1 = Solve(ray, v00, v01, v10, v11, root1, a1, a2, b1, b2, c1, c2, d1, d2);
    float t2 = Solve(ray, v00, v01, v10, v11, root2, a1, a2, b1, b2, c1, c2, d1, d2);

    //return fminf(t1, t2);
    return make_float2(fminf(t1, t2), 2);
  }
  else if(root1 > -EPSILON && root1 < 1 + EPSILON){
    //return 1;

    //return Solve(ray, v00, v01, v10, v11, root1, a1, a2, b1, b2, c1, c2, d1, d2);
    return make_float2(Solve(ray, v00, v01, v10, v11, root1, a1, a2, b1, b2, c1, c2, d1, d2), 2);
  }
  else if(root2 > -EPSILON && root2 < 1 + EPSILON){
    //return 1;

    //return Solve(ray, v00, v01, v10, v11, root2, a1, a2, b1, b2, c1, c2, d1, d2);
    return make_float2(Solve(ray, v00, v01, v10, v11, root2, a1, a2, b1, b2, c1, c2, d1, d2), 2);
  }
  else{
    //return 0;
    //return HIP_INF_F;
    return make_float2(HIP_INF_F, 0);
  }

#else
  
  if(delta < -EPSILON){
    return make_float2(HIP_INF_F, 0);
    //return 0;
    //return HIP_INF_F;
  }
  else if(delta >= -EPSILON && delta <= EPSILON){
    //else if(delta == 0){
    float t = Solve(ray, v00, v01, v10, v11, -aux_b / (2.0f * aux_a), a1, a2, b1, b2, c1, c2, d1, d2);

    if(t + EPSILON < ray->t)
      return make_float2(HIP_INF_F, 1);
    else
      return make_float2(t, 1);
    
    //return make_float2(t, 1);
    //return 1;
    //return t;
  }
  else{
    
    //float v1 = (- aux_b + sqrtf(delta)) / (2.0f * aux_a);
    //float v2 = (- aux_b - sqrtf(delta)) / (2.0f * aux_a);
    float q;
    if(aux_b < 0.0f)
      q = - 0.5f * (aux_b - sqrtf(delta));
    else
      q = - 0.5f * (aux_b + sqrtf(delta));

    float v1 = q / aux_a;
    float v2 = aux_c / q;

    float t1 = Solve(ray, v00, v01, v10, v11, v1, a1, a2, b1, b2, c1, c2, d1, d2);
    float t2 = Solve(ray, v00, v01, v10, v11, v2, a1, a2, b1, b2, c1, c2, d1, d2);
    
    if(t1 + EPSILON < ray->t)
      t1 = HIP_INF_F;
    if(t2 + EPSILON < ray->t)
      t2 = HIP_INF_F;
    

    if(getfart)
      return make_float2(fmaxf(t1, t2), 2);
    else
      return make_float2(fminf(t1, t2), 2);

    /*
    if(t2 == HIP_INF_F)
    return make_float2(t1, 1);
    else if(t1 == HIP_INF_F)
    return make_float2(t2, 1);
    else
    return make_float2(tmin, 2);
    */
    //return 2;


    /*
    if(tmin < HIP_INF_F)
    return tmin;
    else
    return -1;

    if(t1 < HIP_INF_F && t2 < HIP_INF_F)
    return 2;
    else if(t1 < HIP_INF_F)
    return 1;
    else if(t2 < HIP_INF_F)
    return 1;
    else
    return 0;
    */
  }
#endif
  /*
  if(delta == 0){
  float v = -aux_b / (2 * aux_a);
  return Solve(ray, v00, v01, v10, v11, v, a1, a2, b1, b2, c1, c2, d1, d2);
  }
  return 0;
  */

}


/**
* Calculate ZetaPsi, using gaussian quadrature (hexahedral mesh)
*/
__device__ float4 GetZetaPsiQuad(Ray* ray, float4 eyepos, float raySegLength, float alphaBack, float alphaFront, float sback, float sfront){

  float pf = alphaBack;
  float pb = alphaFront;
  float D = raySegLength;
  
  float a1 = eyepos.x;
  float a2 = ray->dir.x;
  float b1 = eyepos.y;
  float b2 = ray->dir.y;
  float g1 = eyepos.z;
  float g2 = ray->dir.z;

  float c0 = ray->currentelem.interpolfunc1.w;
  float c1 = ray->currentelem.interpolfunc0.x;
  float c2 = ray->currentelem.interpolfunc0.y;
  float c3 = ray->currentelem.interpolfunc0.z;
  float c4 = ray->currentelem.interpolfunc0.w;
  float c5 = ray->currentelem.interpolfunc1.x;
  float c6 = ray->currentelem.interpolfunc1.y;
  float c7 = ray->currentelem.interpolfunc1.z;

  float4 w; //{w0, w1, w2, w3}
  w.x = c0 + a1*c1 + b1*c2 + a1*b1*c4 + c3*g1 + a1*c5*g1 + b1*c6*g1 + a1*b1*c7*g1;
  w.y = a2*c1 + b2*c2 + a2*b1*c4 + a1*b2*c4 + a2*c5*g1 + b2*c6*g1 + a2*b1*c7*g1 + a1*b2*c7*g1 + c3*g2 + a1*c5*g2 + b1*c6*g2 + a1*b1*c7*g2;
  w.z = a2*b2*c4 + a2*b2*c7*g1 + a2*c5*g2 + b2*c6*g2 + a2*b1*c7*g2 + a1*b2*c7*g2;
  w.w = a2*b2*c7*g2;
  /*
  w.x = fabs(w.x);
  w.y = fabs(w.y);
  w.z = fabs(w.z);
  w.w = fabs(w.w);
  */

  //w += make_float4(1.0f);

  float zeta = expf(- (6.0f * D * (pb + pf) * w.y + 4.0f * D * D * (pb + 2.0f * pf) * w.z + 3.0f * D * D * D * (pb + 3.0f * pf) * w.w) / (12.0f * w.y + 12.0f * D * w.z + 12.0f * D * D * w.w));

  float aux = (w.y * D + w.z * D * D + w.w * D * D * D);
  //Psi (using gauss quadrature)
  float4 psi1, psi2, exp1, exp2;
  float Doveraux = D / aux;
  
  exp1.x = - (Doveraux * (0.980145 * aux * pf + D * (pb - pf) * (0.499803 * w.y + D * (0.333331 * w.z + 0.25 * D * w.w))));
  exp1.y = - (Doveraux * (0.898333 * aux * pf + D * (pb - pf) * (0.494832 * w.y + D * (0.332983 * w.z + 0.249973 * D * w.w))));
  exp1.z = - (Doveraux * (0.762766 * aux * pf + D * (pb - pf) * (0.47186 * w.y + D * (0.328883 * w.z + 0.249208 * D * w.w))));
  exp1.w = - (Doveraux * (0.591717 * aux * pf + D * (pb - pf) * (0.416653 * w.y + D * (0.310647 * w.z + 0.243053 * D * w.w))));

  exp2.x = - (Doveraux * (0.408283 * aux * pf + D * (pb - pf) * (0.324935 * w.y + D * (0.264274 * w.z + 0.219352 * D * w.w))));
  exp2.y = - (Doveraux * (0.237234 * aux * pf + D * (pb - pf) * (0.209094 * w.y + D * (0.185404 * w.z + 0.165374 * D * w.w))));
  exp2.z = - (Doveraux * (0.101667 * aux * pf + D * (pb - pf) * (0.0964987 * w.y + D * (0.0916809 * w.z + 0.0871867 * D * w.w))));
  exp2.w = - (Doveraux * (0.0198551 * aux * pf + D * (pb - pf) * (0.019658 * w.y + D * (0.0194635 * w.z + 0.0192715 * D * w.w))));

  psi1.x = 0.0506143 * D * expf(exp1.x) * (w.y + 0.0397101 * D * w.z + 0.00118267 * D * D * w.w);
  psi1.y = 0.111191 * D * expf(exp1.y) * (w.y + 0.203334 * D * w.z + 0.0310084 * D * D * w.w);
  psi1.z = 0.156853 * D * expf(exp1.z) * (w.y + 0.474468 * D * w.z + 0.16884 * D * D * w.w);
  psi1.w = 0.181342 * D * expf(exp1.w) * (w.y + 0.816565 * D * w.z + 0.500084 * D * D * w.w);

  psi2.x = 0.181342 * D * expf(exp2.x) * (w.y + 1.18343 * D * w.z + 1.05039 * D * D * w.w);
  psi2.y = 0.156853 * D * expf(exp2.y) * (w.y + 1.52553 * D * w.z + 1.74544 * D * D * w.w);
  psi2.z = 0.111191 * D * expf(exp2.z) * (w.y + 1.79667 * D * w.z + 2.42101  * D * D * w.w);
  psi2.w = 0.0506143 * D * expf(exp2.w) * (w.y + 1.96029 * D * w.z + 2.88205 * D * D * w.w);
  
  /*
  psi1.x = 0.0506143 * expf(- Doveraux * (0.980145 * aux * pf + 0.084 * (pb - pf) * (5.99763 * D * w.y + 3.99997 * D * D * w.z - 3.0f * D * D * D * w.w))) * (w.y + 0.0397101 * w.z + 0.00118267 * w.w);
  psi1.y = 0.111191 * expf(- Doveraux * (0.898333 * aux * pf + 0.084 * (pb - pf) * (5.93798 * D * w.y + 3.9958 * D * D * w.z - 2.99968 * D * D * D * w.w))) * (w.y + 0.203334 * w.z + 0.0310084 * w.w);
  psi1.z = 0.156853 * expf(- Doveraux * (0.762766  * aux * pf + 0.084 * (pb - pf) * (5.66232 * D * w.y + 3.94659 * D * D * w.z - 2.9905 * D * D * D * w.w))) * (w.y + 0.474468 * w.z + 0.16884 * w.w);
  psi1.w = 0.181342 * expf(- Doveraux * (0.591717 * aux * pf + 0.084 * (pb - pf) * (4.99983 * D * w.y + 3.72777 * D * D * w.z - 2.91664 * D * D * D * w.w))) * (w.y + 0.816565 * w.z + 0.500084 * w.w);

  psi2.x = 0.181342 * expf(- Doveraux * (0.408283 * aux * pf + 0.084 * (pb - pf) * (3.89922  * D * w.y + 3.17129 * D * D * w.z - 2.63223 * D * D * D * w.w))) * (w.y + 1.18343 * w.z + 1.05039 * w.w);
  psi2.y = 0.156853 * expf(- Doveraux * (0.237234  * aux * pf + 0.084 * (pb - pf) * (2.50913  * D * w.y + 2.22485 * D * D * w.z - 1.98448 * D * D * D * w.w))) * (w.y + 1.52553 * w.z + 1.74544 * w.w);
  psi2.z = 0.111191 * expf(- Doveraux * (0.101667 * aux * pf + 0.084 * (pb - pf) * (1.15798  * D * w.y + 1.10017 * D * D * w.z - 1.04624 * D * D * D * w.w))) * (w.y + 1.79667 * w.z + 2.42101  * w.w);
  psi2.w = 0.0506143 * expf(- Doveraux * (0.0198551 * aux * pf + 0.084 * (pb - pf) * (0.235896 * D * w.y + 0.233561 * D * D * w.z - 0.231258 * D * D * D * w.w))) * (w.y + 1.96029 * w.z + 2.88205 * w.w);
  */
  float psi = (psi1.x + psi1.y + psi1.z + psi1.w + psi2.x + psi2.y + psi2.z + psi2.w);

  return make_float4(zeta, psi , psi, psi);
}

/**
* Calculate ZetaPsi, fetching psi gamma from texture (hexahedral mesh)
*/
__device__ float4 GetZetaPsiFetch(Ray* ray, float4 eyepos, float raySegLength, float alphaBack, float alphaFront, float sback, float sfront){
  /*
  float a1 = (eyepos.x+(ray->t)*ray->dir.x);
  float a2 = ray->dir.x;
  float b1 = (eyepos.y+(ray->t)*ray->dir.y);
  float b2 = ray->dir.y;
  float g1 = (eyepos.z+(ray->t)*ray->dir.z);
  float g2 = ray->dir.z;

  float c1 = ray->currentelem.interpolfunc1.w;
  float c2 = ray->currentelem.interpolfunc0.x;
  float c3 = ray->currentelem.interpolfunc0.y;
  float c4 = ray->currentelem.interpolfunc0.z;
  float c5 = ray->currentelem.interpolfunc0.w;
  float c6 = ray->currentelem.interpolfunc1.x;
  float c7 = ray->currentelem.interpolfunc1.y;
  float c8 = ray->currentelem.interpolfunc1.z;


  float4 w;
  w.x = c1 + a1*c2 + b1*c3 + a1*b1*c5 + c4*g1 + a1*c6*g1 + b1*c7*g1 + a1*b1*c8*g1;
  w.y = a2*c2 + b2*c3 + a2*b1*c5 + a1*b2*c5 + a2*c6*g1 + b2*c7*g1 + a2*b1*c8*g1 + a1*b2*c8*g1 + c4*g2 + a1*c6*g2 + b1*c7*g2 + a1*b1*c8*g2;
  w.z = a2*b2*c5 + a2*b2*c8*g1 + a2*c6*g2 + a2*c7*g2 + a2*b1*c8*g2 + a1*b2*c8*g2;
  w.w = a2*b2*c8*g2;

  float polyn = w.y + raySegLength * w.z + raySegLength * raySegLength * w.w;
  float3 num = (alphaBack - alphaFront) * make_float3(w.y, w.z, w.w) / polyn;

  float3 gamma = num / (num + 1.0f);
  float4 zetapsigamma = tex3D(texZetaPsiGamma, gamma.x, gamma.y, gamma.z) * expf(- raySegLength * alphaFront);
  zetapsigamma.y *= raySegLength;
  zetapsigamma.z *= raySegLength;
  zetapsigamma.w *= raySegLength;

  float psi = ((((w.y * zetapsigamma.y)) + ((2 * w.z * zetapsigamma.z)) + ((3 * w.w * zetapsigamma.w))) / polyn);

  return make_float4(zetapsigamma.x, psi, psi, psi);
  */
  return make_float4(0, 0, 0, 0);
}

/**
* Find scalar of the (x,y,z) point (hexahedral mesh)
*/
__device__ float FindScalar(Ray* ray, float p_t){

  float4 pos = ray->eyepos + p_t * ray->dir;
  pos.w = 1.0;

  float4 interpolfunc0 = ray->currentelem.interpolfunc0;
  float4 interpolfunc1 = ray->currentelem.interpolfunc1;

  return interpolfunc0.x * pos.x + interpolfunc0.y * pos.y + interpolfunc0.z * pos.z + interpolfunc0.w * pos.x * pos.y
    + interpolfunc1.x * pos.x * pos.z + interpolfunc1.y * pos.y * pos.z + interpolfunc1.z * pos.x * pos.y * pos.z + interpolfunc1.w;


}

/**
* Find integration step (hex)
*/
/*
inline __device__ float FindIntegrationStep(Ray* ray, float t, float diffcpfront, float diffbackfront){

  float c0 = ray->currentelem.interpolfunc1.w;
  float c1 = ray->currentelem.interpolfunc0.x;
  float c2 = ray->currentelem.interpolfunc0.y;
  float c3 = ray->currentelem.interpolfunc0.z;
  float c4 = ray->currentelem.interpolfunc0.w;
  float c5 = ray->currentelem.interpolfunc1.x;
  float c6 = ray->currentelem.interpolfunc1.y;
  float c7 = ray->currentelem.interpolfunc1.z;
  
  float ox = eyepos.x;
  float oy = eyepos.y;
  float oz = eyepos.z;

  float dx = threadray->dir.x;
  float dy = threadray->dir.y;
  float dz = threadray->dir.z;

  float a = c0 + c1*ox + c2*oy + c4*ox*oy + c3*oz + c6*ox*oz + c5*oy*oz + c7*ox*oy*oz;
  float b = c1*dx + c2*dy + c3*dz + c4*dy*ox + c6*dz*ox + c4*dx*oy + c5*dz*oy + c7*dz*ox*oy + c6*dx*oz + c5*dy*oz + c7*dy*ox*oz + c7*dx*oy*oz;
  float c = c4*dx*dy + c6*dx*dz + c5*dy*dz + c7*dy*dz*ox + c7*dx*dz*oy + c7*dx*dy*oz;
  float d = c7*dx*dy*dz - isoScalar;


  float delta = 18 * a * b * c * d + 4 * b * b * b * d + b * b + c * c - 4 * a * c * c * c - 27 * a * a * d * d;
  
}
*/

#else
/**
* Calculate ZetaPsi, using gaussian quadrature (tetrahedral mesh)
*/
__device__ float4 GetZetaPsiQuad(Ray* ray, float4 eyepos, float raySegLength, float alphaBack, float alphaFront, float sback, float sfront){
  float4 t1, weights1, expf_psi1, t2, weights2, expf_psi2;
  float psi;
  float2 alphaL; // alpha * rayLength
  alphaL = raySegLength * make_float2(alphaBack, alphaFront);

  //Zeta
  float zeta = expf(-dot(alphaL, make_float2(0.5f, 0.5f)));

  //Psi
  t1.x = 0.019855071751231912;
  t1.y = 0.10166676129318664;
  t1.z = 0.2372337950418355;
  t1.w = 0.40828267875217505;
  t2.x = 0.591717321247825;
  t2.y = 0.7627662049581645;
  t2.z = 0.8983332387068134;
  t2.w = 0.9801449282487681;
  weights1.x = 0.05061426814518863;
  weights1.y = 0.11119051722668714;
  weights1.z = 0.15685332293894347;
  weights1.w = 0.18134189168918077;
  weights2.x = 0.18134189168918077;
  weights2.y = 0.15685332293894347;
  weights2.z = 0.11119051722668714;
  weights2.w = 0.05061426814518863;

  expf_psi1 = - make_float4(raySegLength) * ((t1) * (alphaBack * (make_float4(1)-t1) + alphaFront * t1));
  expf_psi2 = - make_float4(raySegLength) * ((t2) * (alphaBack * (make_float4(1)-t2) + alphaFront * t2));

  psi = dot(make_float4(expf(expf_psi1.x), expf(expf_psi1.y), expf(expf_psi1.z), expf(expf_psi1.w)), weights1);  
  psi += dot(make_float4(expf(expf_psi2.x), expf(expf_psi2.y), expf(expf_psi2.z), expf(expf_psi2.w)), weights2);

  return make_float4(zeta, psi, psi, psi);
}


/**
* Calculate ZetaPsi, fetching psi gamma from texture (tetrahedral mesh)
*/
__device__ float4 GetZetaPsiFetch(Ray* ray, float4 eyepos, float raySegLength, float alphaBack, float alphaFront, float sback, float sfront){

  float2 alphaL; // alpha * rayLength
  alphaL = raySegLength * make_float2(alphaBack, alphaFront);

  //Zeta
  float zeta = expf(-dot(alphaL, make_float2(0.5f, 0.5f)));

  //Gamma
  float2 gamma = alphaL / (alphaL + make_float2(1.0f));

  //Psi
  float psi = tex2D(texZetaPsiGamma, gamma.x, gamma.y);
  return make_float4(zeta, psi, psi, psi);
}

/**
* Find scalar of the (x,y,z) point (tetrahedral mesh)
*/
inline __device__ float FindScalar(Ray* ray, float p_t){

  float4 pos = ray->eyepos + p_t * ray->dir;
  pos.w = 1.0;

  return dot(pos, ray->currentelem.interpolfunc0);
}

/**
* Find integration step (tet)
*/
inline __device__ float FindIntegrationStep(Ray* ray, float t, float diffcpfront, float diffbackfront){
  return (ray->t) + (t - ray->t) * (diffcpfront / diffbackfront);
  //t = (isoScalar - dot(threadRay->eyepos, threadRay->currentelem.interpolfunc0) - threadRay->currentelem.interpolfunc0.w) / (dot(threadRay->dir, threadRay->currentelem.interpolfunc0));
}

#endif

/**
* Find control point
*/
inline __device__ float FindControlPoint(Ray* ray, float backscalar, float4 cpvalues){

  float cpscalar;
  float cpnextscalar;
  
  if(ray->frontscalar > backscalar){
    cpscalar = cpvalues.z;
    cpnextscalar = cpvalues.w;

    if(ray->frontscalar <= cpscalar)
      cpscalar = cpnextscalar;
  }
  else{
    cpscalar = cpvalues.x;
    cpnextscalar = cpvalues.y;

    if(ray->frontscalar >= cpscalar)
      cpscalar = cpnextscalar;
  }

  return cpscalar;
}



/**
* Constant integration of the ray
*/
__device__ void IntegrateRayConst(Ray* ray, float4 eyepos, float raySegLength, float sback, float sfront){

  float4 avg = tex1D(texVolumetricColorScale, 0.5*(sback+sfront));
  //float4 avg = tex1D(texColorScale, sback);
  float zeta = expf(- raySegLength * avg.w);

  float alpha = 1 - zeta;
#ifdef CUDARC_WHITE
  float3 color = (make_float3(1) - make_float3(avg)) * alpha;
  ray->acccolor += (1 - ray->acccolor.w) * make_float4(-color.x, -color.y, -color.z, alpha);
#else
  float3 color = (make_float3(avg)) * alpha;
  ray->acccolor += (1 - ray->acccolor.w) * make_float4(color.x, color.y, color.z, alpha);
#endif
}


/**
* Linear integration/trilinear of the ray
*/
__device__ void IntegrateRayLinear(Ray* ray, float4 eyepos, float raySegLength, float sback, float sfront){

  float3 color;
  float alpha;

  float4 colorBack = tex1D(texVolumetricColorScale, sback);
  float4 colorFront = tex1D(texVolumetricColorScale, sfront);
  float4 zetapsi;

  if(constMemory.interpoltype == Quad)
    zetapsi = GetZetaPsiQuad(ray, eyepos, raySegLength, colorBack.w, colorFront.w, sback, sfront);
  else
    zetapsi = GetZetaPsiFetch(ray, eyepos, raySegLength, colorBack.w, colorFront.w, sback, sfront);

  alpha = 1 - zetapsi.x;
  //Finally
#ifdef CUDARC_HEX
#ifdef CUDARC_WHITE
  color = (make_float3(1)- make_float3(colorFront)) * (zetapsi.y - zetapsi.x) + (make_float3(1) - make_float3(colorBack)) * (1.0f - zetapsi.y);
  ray->acccolor += (1 - ray->acccolor.w) * make_float4(-color.x, -color.y, -color.z, alpha);
#else
  color = make_float3(colorFront) * (zetapsi.y - zetapsi.x) + make_float3(colorBack) * (1.0f - zetapsi.y);
  ray->acccolor += (1 - ray->acccolor.w) * make_float4(color.x, color.y, color.z, alpha);
#endif
#else
#ifdef CUDARC_WHITE
  color = (make_float3(1) - make_float3(colorBack)) * (zetapsi.y - zetapsi.x) + (make_float3(1)- make_float3(colorFront)) * (1.0f - zetapsi.y);
  ray->acccolor += (1 - ray->acccolor.w) * make_float4(-color.x, -color.y, -color.z, alpha);
#else
  color = (make_float3(colorBack)) * (zetapsi.y - zetapsi.x) + (make_float3(colorFront)) * (1.0f - zetapsi.y);
  ray->acccolor += (1 - ray->acccolor.w) * make_float4(color.x, color.y, color.z, alpha);
#endif
#endif
  /*
  if(zetapsi.y < 0)
    ray->acccolor = make_float4(1, 0, 0, 1);
  else if(zetapsi.y > 1.0f)
    ray->acccolor = make_float4(0, 1, 0, 1);
  else if(zetapsi.y == 0.0f)
    ray->acccolor = make_float4(1, 0, 1, 1);
  else if(zetapsi.y == 1.0f)
    ray->acccolor = make_float4(0, 1, 1, 1);
  else
    ray->acccolor = make_float4(0, 0, 1, 1);
  */
  /*
  if(sback == sfront)
    ray->acccolor = make_float4(1, 0, 0, 1);
  else
    ray->acccolor = make_float4(0, 0, 1, 1);
  */
  
}


/**
* Initialize function, calculate the starting position of the ray on the mesh
*/
__device__ Ray Initialize(int x, int y, int offset, float4 eyePos){

  float4 tetraInfo = tex2D(texIntersect, x, y);
  float4 dir = make_float4(tetraInfo.x, tetraInfo.y, tetraInfo.z, 0);
  int tid = floor(tetraInfo.w + 0.5f);

  Ray threadRay;
  threadRay.t = 0.0f;
  threadRay.dir = normalize(dir);
  threadRay.eyepos = eyePos;
#ifdef CUDARC_HEX
  threadRay.frontid = tid / 6; 
  threadRay.frontface = tid % 6; 
#else
  threadRay.frontid = tid / 4;  
  threadRay.frontface = tid % 4;  
#endif
#ifdef CUDARC_WHITE
  threadRay.acccolor = make_float4(1, 1, 1, 0);
#else
  threadRay.acccolor = make_float4(0);
#endif

  threadRay.currentelem.interpolfunc0 = tex1Dfetch(texInterpolFunc0, threadRay.frontid);
#ifdef CUDARC_HEX
  threadRay.currentelem.interpolfunc1 = tex1Dfetch(texInterpolFunc1, threadRay.frontid);
#endif


#ifdef CUDARC_BILINEAR
  float4 v0 = tex1Dfetch(texNode0, threadRay.frontid);
  float4 v1 = tex1Dfetch(texNode1, threadRay.frontid);
  float4 v2 = tex1Dfetch(texNode2, threadRay.frontid);
  float4 v3 = tex1Dfetch(texNode3, threadRay.frontid);
  float4 v4 = tex1Dfetch(texNode4, threadRay.frontid);
  float4 v5 = tex1Dfetch(texNode5, threadRay.frontid);
  float4 v6 = tex1Dfetch(texNode6, threadRay.frontid);
  float4 v7 = tex1Dfetch(texNode7, threadRay.frontid);

  if(threadRay.frontface == 0)
    threadRay.t = Intersect(&threadRay, v5, v4, v6, v7, false).x; //fem
    //threadRay.t = Intersect(&threadRay, v0, v1, v2, v3, false).x; //res
  else if(threadRay.frontface == 1)
    threadRay.t = Intersect(&threadRay, v0, v1, v3, v2, false).x; //fem
    //threadRay.t = Intersect(&threadRay, v4, v5, v6, v7, false).x; //res
  else if(threadRay.frontface == 2)
    threadRay.t = Intersect(&threadRay, v1, v2, v5, v6, false).x; //fem
    //threadRay.t = Intersect(&threadRay, v1, v3, v5, v7, false).x; //res
  else if(threadRay.frontface == 3)'
    threadRay.t = Intersect(&threadRay, v0, v3, v4, v7, false).x; //fem
    //threadRay.t = Intersect(&threadRay, v0, v2, v4, v6, false).x; //res
  else if(threadRay.frontface == 4)
    threadRay.t = Intersect(&threadRay, v0, v1, v4, v5, false).x; //fem
    //threadRay.t = Intersect(&threadRay, v0, v1, v4, v5, false).x; //res
  else if(threadRay.frontface == 5)
    threadRay.t = Intersect(&threadRay, v2, v3, v6, v7, false).x; //fem
    //threadRay.t = Intersect(&threadRay, v2, v3, v6, v7, false).x; //res

  if(threadRay.t == HIP_INF_F)
    threadRay.acccolor = make_float4(0,0,0,1);

  //threadRay.t = length(dir);

#else
   threadRay.t = length(dir);
#endif

  threadRay.frontscalar = FindScalar(&threadRay, threadRay.t);
  /*
  if(threadRay.t == HIP_INF_F)
    threadRay.acccolor = make_float4(1, 0, 0, 1);
  else
    threadRay.acccolor = make_float4(0, 0, 1, 1);
  */

  return threadRay;

}

/**
* Volumetric traverse the ray through the mesh
*/
__device__ void Traverse(int x, int y, int offset, Ray* threadRay){

  float4 planeEq;
  float sameDirection;
  float t = HIP_INF_F;
  int backid = 0;
  int backfaceid = 0;
  float round = 0;

#ifdef CUDARC_HEX
  float4 hexAdj1 = tex1Dfetch(texAdj0, threadRay->frontid);
  float4 hexAdj2 = tex1Dfetch(texAdj1, threadRay->frontid);
#else
  float4 tetAdj = tex1Dfetch(texAdj0, threadRay->frontid);
#endif

#ifdef CUDARC_HEX
  float4 v0, v1, v2, v3, v4, v5, v6, v7;
  float4 ray = cross(threadRay->dir, threadRay->eyepos);
  /*
  float4 v0 = tex1Dfetch(texNode0, threadRay->frontid);
  float4 v1 = tex1Dfetch(texNode1, threadRay->frontid);
  float4 v2 = tex1Dfetch(texNode2, threadRay->frontid);
  float4 v3 = tex1Dfetch(texNode3, threadRay->frontid);
  float4 v4 = tex1Dfetch(texNode4, threadRay->frontid);
  float4 v5 = tex1Dfetch(texNode5, threadRay->frontid);
  float4 v6 = tex1Dfetch(texNode6, threadRay->frontid);
  float4 v7 = tex1Dfetch(texNode7, threadRay->frontid);
  float4 ray = cross(threadRay->dir, threadRay->eyepos);
  float4 point;
  
  float4 v02 = (v0 - v2);
  float4 q02 = cross(v02, v0);
  float ps02 = permuted_inner_produtct(v02, q02, threadRay->dir, ray);

  float4 v32 = (v3 - v2);
  float4 q32 = cross(v32, v2);
  float ps32 = permuted_inner_produtct(v32, q32, threadRay->dir, ray);

  float4 v13 = (v1 - v3);
  float4 q13 = cross(v13, v1);
  float ps13 = permuted_inner_produtct(v13, q13, threadRay->dir, ray);

  float4 v01 = (v0 - v1);
  float4 q01 = cross(v01, v0);
  float ps01 = permuted_inner_produtct(v01, q01, threadRay->dir, ray);

  float4 v21 = (v2 - v1);
  float4 q21 = cross(v21, v1);
  float ps21 = permuted_inner_produtct(v21, q21, threadRay->dir, ray);

  float4 v76 = (v7 - v6);
  float4 q76 = cross(v76, v6);
  float ps76 = permuted_inner_produtct(v76, q76, threadRay->dir, ray);

  float4 v57 = (v5 - v7);
  float4 q57 = cross(v57, v7);
  float ps57 = permuted_inner_produtct(v57, q57, threadRay->dir, ray);

  float4 v46 = (v4 - v6);
  float4 q46 = cross(v46, v6);
  float ps46 = permuted_inner_produtct(v46, q46, threadRay->dir, ray);

  float4 v45 = (v4 - v5);
  float4 q45 = cross(v45, v5);
  float ps45 = permuted_inner_produtct(v45, q45, threadRay->dir, ray);

  float4 v26 = (v2 - v6);
  float4 q26 = cross(v26, v6);
  float ps26 = permuted_inner_produtct(v26, q26, threadRay->dir, ray);

  float4 v37 = (v3 - v7);
  float4 q37 = cross(v37, v7);
  float ps37 = permuted_inner_produtct(v37, q37, threadRay->dir, ray);

  float4 v04 = (v0 - v4);
  float4 q04 = cross(v04, v4);
  float ps04 = permuted_inner_produtct(v04, q04, threadRay->dir, ray);

  float4 v15 = (v1 - v5);
  float4 q15 = cross(v15, v5);
  float ps15 = permuted_inner_produtct(v15, q15, threadRay->dir, ray);
  */
#else
#ifdef CUDARC_PLUCKER
  float4 v0 = tex1Dfetch(texNode0, threadRay->frontid);
  float4 v1 = tex1Dfetch(texNode1, threadRay->frontid);
  float4 v2 = tex1Dfetch(texNode2, threadRay->frontid);
  float4 v3 = tex1Dfetch(texNode3, threadRay->frontid);
  float4 ray = cross(threadRay->dir, threadRay->eyepos);
  float4 point;

  float4 v02 = (v0 - v2);
  float4 q02 = cross(v02, v0);
  float ps02 = permuted_inner_produtct(v02, q02, threadRay->dir, ray);

  float4 v32 = (v3 - v2);
  float4 q32 = cross(v32, v2);
  float ps32 = permuted_inner_produtct(v32, q32, threadRay->dir, ray);

  float4 v03 = (v0 - v3);
  float4 q03 = cross(v03, v0);
  float ps03 = permuted_inner_produtct(v03, q03, threadRay->dir, ray);

  float4 v13 = (v1 - v3);
  float4 q13 = cross(v13, v1);
  float ps13 = permuted_inner_produtct(v13, q13, threadRay->dir, ray);

  float4 v01 = (v0 - v1);
  float4 q01 = cross(v01, v0);
  float ps01 = permuted_inner_produtct(v01, q01, threadRay->dir, ray);

  float4 v21 = (v2 - v1);
  float4 q21 = cross(v21, v1);
  float ps21 = permuted_inner_produtct(v21, q21, threadRay->dir, ray);
#endif
#endif


  int aux = 0;
  while((constMemory.numtraverses > 0 && constMemory.debug) || (threadRay->frontid > 0 && threadRay->acccolor.w < 0.99)){
    
    if((constMemory.numtraverses > 0 && aux >= constMemory.numtraverses) || aux >= CUDARC_MAX_ITERATIONS)
      break;

    aux++;

    //t = -(P0 . N + d) / (V . N) (http://www.cs.princeton.edu/courses/archive/fall00/cs426/lectures/raycast/sld017.htm)
    //t = -(eyePos . normal + d) / (eyeDir . normal)
    threadRay->dir.w = 0;
    threadRay->eyepos.w = 1;

#ifdef CUDARC_HEX
#ifdef CUDARC_BILINEAR
    v0 = tex1Dfetch(texNode0, threadRay->frontid);
    v1 = tex1Dfetch(texNode1, threadRay->frontid);
    v2 = tex1Dfetch(texNode2, threadRay->frontid);
    v3 = tex1Dfetch(texNode3, threadRay->frontid);
    v4 = tex1Dfetch(texNode4, threadRay->frontid);
    v5 = tex1Dfetch(texNode5, threadRay->frontid);
    v6 = tex1Dfetch(texNode6, threadRay->frontid);
    v7 = tex1Dfetch(texNode7, threadRay->frontid);


    //Ray Bilinear patch intersection
    float2 t0 = make_float2(HIP_INF_F, 0);
    float2 t1 = make_float2(HIP_INF_F, 0);
    float2 t2 = make_float2(HIP_INF_F, 0);
    float2 t3 = make_float2(HIP_INF_F, 0);
    float2 t4 = make_float2(HIP_INF_F, 0);
    float2 t5 = make_float2(HIP_INF_F, 0);

    
    ///res
    /*
    t0.x = Intersect(threadRay, v0, v1, v2, v3, threadRay->frontface == 0 ? 1 : 0).x;
    t1.x = fminf(t, Intersect(threadRay, v4, v5, v6, v7, threadRay->frontface == 1 ? 1 : 0).x);
    t2.x = fminf(t, Intersect(threadRay, v1, v3, v5, v7, threadRay->frontface == 2 ? 1 : 0).x);
    t3.x = fminf(t, Intersect(threadRay, v0, v2, v4, v6, threadRay->frontface == 3 ? 1 : 0).x);
    t4.x = fminf(t, Intersect(threadRay, v2, v3, v6, v7, threadRay->frontface == 5 ? 1 : 0).x);
    t5.x = fminf(t, Intersect(threadRay, v0, v1, v4, v5, threadRay->frontface == 4 ? 1 : 0).x);
    */
    
    //fem
    
    t0.x = Intersect(threadRay, v0, v1, v3, v2, threadRay->frontface == 1 ? 1 : 0).x;
    t1.x = fminf(t, Intersect(threadRay, v5, v4, v6, v7, threadRay->frontface == 0 ? 1 : 0).x);
    t2.x = fminf(t, Intersect(threadRay, v1, v2, v5, v6, threadRay->frontface == 2 ? 1 : 0).x);
    t3.x = fminf(t, Intersect(threadRay, v0, v3, v4, v7, threadRay->frontface == 3 ? 1 : 0).x);
    t4.x = fminf(t, Intersect(threadRay, v2, v3, v6, v7, threadRay->frontface == 5 ? 1 : 0).x);
    t5.x = fminf(t, Intersect(threadRay, v0, v1, v4, v5, threadRay->frontface == 4 ? 1 : 0).x);
    

    if(t0.x < t1.x && t0.x < t2.x && t0.x < t3.x && t0.x < t4.x && t0.x < t5.x){
      t = t0.x;
      //round = hexAdj1.y; //fem
      round = hexAdj1.x; //res
      //round = 0;
      if(constMemory.debug) threadRay->acccolor = make_float4(1, 0, 0, 1);
    }
    if(t1.x < t0.x && t1.x < t2.x && t1.x < t3.x && t1.x < t4.x && t1.x < t5.x){
      t = t1.x;
      //round = hexAdj1.x; //fem
      round = hexAdj1.y; //res
      //round = 0;
      if(constMemory.debug) threadRay->acccolor = make_float4(0, 1, 0, 1);
    }
    if(t2.x < t0.x && t2.x < t1.x && t2.x < t3.x && t2.x < t4.x && t2.x < t5.x){
      t = t2.x;
      //round = hexAdj1.z; //fem
      round = hexAdj1.z; //res
      //round = 0;
      if(constMemory.debug) threadRay->acccolor = make_float4(0, 1, 1, 1);
    }
    if(t3.x < t0.x && t3.x < t1.x && t3.x < t2.x && t3.x < t4.x && t3.x < t5.x){
      t = t3.x;
      //round = hexAdj2.x; //fem
      round = hexAdj2.x; //res
      //round = 0;
      if(constMemory.debug) threadRay->acccolor = make_float4(0, 0, 1, 1);
    }
    if(t4.x < t0.x && t4.x < t1.x && t4.x < t2.x && t4.x < t3.x && t4.x < t5.x){
      t = t4.x;
      //round = hexAdj2.z; //fem
      round = hexAdj2.z; //res
      //round = 0;
      if(constMemory.debug) threadRay->acccolor = make_float4(1, 1, 0, 1);
    }
    if(t5.x < t0.x && t5.x < t1.x && t5.x < t2.x && t5.x < t3.x && t5.x < t4.x){
      t = t5.x;
      //round = hexAdj2.y; //fem
      round = hexAdj2.y; //res
      //round = 0;
      if(constMemory.debug) threadRay->acccolor = make_float4(1, 0, 1, 1);
    }

    if(constMemory.debug) return;

#else

    /*Face 0*/
    if(threadRay->frontface != 0){
      //Triangle 0
      planeEq = tex1Dfetch(texFace0Eq, threadRay->frontid);
      sameDirection = dot(threadRay->dir, planeEq);
      if(sameDirection > 0){
        sameDirection = - dot(planeEq, threadRay->eyepos) / sameDirection;
        if(sameDirection < t){
          t = sameDirection;
          if(constMemory.debug > 0) threadRay->acccolor = make_float4(1, 0, 0, 1);
          //threadRay->accColor = make_float4(0.5, 0.5, 0.5, 1);
          round = hexAdj1.x;
        }
      }
    }

    /*Face 1*/
    if(threadRay->frontface != 1){
      //Triangle 0
      planeEq = tex1Dfetch(texFace1Eq, threadRay->frontid);
      sameDirection = dot(threadRay->dir, planeEq);
      if(sameDirection > 0){
        sameDirection = - dot(planeEq, threadRay->eyepos) / sameDirection;
        if(sameDirection < t){
          t = sameDirection;
          if(constMemory.debug > 0) threadRay->acccolor = make_float4(0, 1, 0, 1);
          //threadRay->accColor = make_float4(0.5, 0.5, 0.5, 1);
          round = hexAdj1.y;
        }
      }
    }

    /*Face 2*/
    if(threadRay->frontface != 2){
      //Triangle 0
      planeEq = tex1Dfetch(texFace2Eq, threadRay->frontid);
      sameDirection = dot(threadRay->dir, planeEq);
      if(sameDirection > 0){
        sameDirection = - dot(planeEq, threadRay->eyepos) / sameDirection;
        if(sameDirection < t){
          t = sameDirection;
          if(constMemory.debug > 0) threadRay->acccolor = make_float4(0, 0, 1, 1);
          //threadRay->accColor = make_float4(0.5, 0.5, 0.5, 1);
          round = hexAdj1.z;
        }
      }
    }

    /*Face 3*/
    if(threadRay->frontface != 3){
      //Triangle 0
      planeEq = tex1Dfetch(texFace3Eq, threadRay->frontid);
      sameDirection = dot(threadRay->dir, planeEq);
      if(sameDirection > 0){
        sameDirection = - dot(planeEq, threadRay->eyepos) / sameDirection;
        if(sameDirection < t){
          t = sameDirection;
          if(constMemory.debug > 0) threadRay->acccolor = make_float4(1, 1, 0, 1);
          //threadRay->accColor = make_float4(0.5, 0.5, 0.5, 1);
          round = hexAdj2.x;
        }
      }
    }

    /*Face 4*/
    if(threadRay->frontface != 4){
      //Triangle 0
      planeEq = tex1Dfetch(texFace4Eq, threadRay->frontid);
      sameDirection = dot(threadRay->dir, planeEq);
      if(sameDirection > 0){
        sameDirection = - dot(planeEq, threadRay->eyepos) / sameDirection;
        if(sameDirection < t){
          t = sameDirection;
          if(constMemory.debug > 0) threadRay->acccolor = make_float4(1, 0, 1, 1);
          //threadRay->accColor = make_float4(0.5, 0.5, 0.5, 1);
          round = hexAdj2.y;
        }
      }
    }

    /*Face 5*/
    if(threadRay->frontface != 5){
      //Triangle 0
      planeEq = tex1Dfetch(texFace5Eq, threadRay->frontid);
      sameDirection = dot(threadRay->dir, planeEq);
      if(sameDirection > 0){
        sameDirection = - dot(planeEq, threadRay->eyepos) / sameDirection;
        if(sameDirection < t){
          t = sameDirection;
          if(constMemory.debug > 0) threadRay->acccolor = make_float4(0, 1, 1, 1);
          //threadRay->accColor = make_float4(0.5, 0.5, 0.5, 1);
          round = hexAdj2.z;
        }
      }
    }




#endif
#else
#ifdef CUDARC_PLUCKER

    v0 = tex1Dfetch(texNode0, threadRay->frontid);
    v1 = tex1Dfetch(texNode1, threadRay->frontid);
    v2 = tex1Dfetch(texNode2, threadRay->frontid);
    v3 = tex1Dfetch(texNode3, threadRay->frontid);

    v02 = (v0 - v2);
    q02 = cross(v02, v0);
    ps02 = permuted_inner_produtct(v02, q02, threadRay->dir, ray);

    v32 = (v3 - v2);
    q32 = cross(v32, v2);
    ps32 = permuted_inner_produtct(v32, q32, threadRay->dir, ray);

    v03 = (v0 - v3);
    q03 = cross(v03, v0);
    ps03 = permuted_inner_produtct(v03, q03, threadRay->dir, ray);

    v13 = (v1 - v3);
    q13 = cross(v13, v1);
    ps13 = permuted_inner_produtct(v13, q13, threadRay->dir, ray);

    v01 = (v0 - v1);
    q01 = cross(v01, v0);
    ps01 = permuted_inner_produtct(v01, q01, threadRay->dir, ray);

    v21 = (v2 - v1);
    q21 = cross(v21, v1);
    ps21 = permuted_inner_produtct(v21, q21, threadRay->dir, ray);


    //Plucker tests
    if(threadRay->frontface == 0.0f){
      //Face 0: 1, 2, 3
      /*
      v0 = tex1Dfetch(texNode0, threadRay->frontid);

      v01 = (v0 - v1);
      q01 = cross(v01, v0);
      ps01 = permuted_inner_produtct(v01, q01, threadRay->dir, ray);

      v02 = (v0 - v2);
      q02 = cross(v02, v0);
      ps02 = permuted_inner_produtct(v02, q02, threadRay->dir, ray);

      v03 = (v0 - v3);
      q03 = cross(v03, v0);
      ps03 = permuted_inner_produtct(v03, q03, threadRay->dir, ray);

      v21 = - v21;
      v13 = - v13;
      v32 = - v32;
      */
      //Test against faces 1, 2, 3
      if((-ps32 <= 0 && -ps03 <= 0 && ps02 <= 0)){
        float3 u = make_float3(-ps32, -ps03, ps02) / (-ps32 -ps03 + ps02);
        if(constMemory.debug) threadRay->acccolor = make_float4(0, 1, 0, 1);
        round = tetAdj.y;
        point = u.x * v0 +  u.y * v2 + u.z * v3;
        point.w = 1.0f;
      }
      else{
        if((-ps13 <= 0 && -ps01 <= 0 && ps03 <= 0)){
          float3 u = make_float3(-ps13, -ps01, ps03) / (-ps13 -ps01 + ps03);
          if(constMemory.debug) threadRay->acccolor = make_float4(0, 0, 1, 1);
          round = tetAdj.z;
          point = u.x * v0 +  u.y * v3 + u.z * v1;
          point.w = 1.0f;
        }
        else{
          if((-ps21 <= 0 && -ps02 <= 0 && ps01 <= 0)){
            float3 u = make_float3(-ps21, -ps02, ps01) / (-ps21 -ps02 + ps01);
            if(constMemory.debug) threadRay->acccolor = make_float4(1, 0, 1, 1);
            round = tetAdj.w;
            point = u.x * v0 +  u.y * v1 + u.z * v2;
            point.w = 1.0f;
          }
        } 
      }
    }
    else if(threadRay->frontface == 1.0f){
      //Face 1: 2, 0, 3

      /*
      v1 = tex1Dfetch(texNode1, threadRay->frontid);

      v13 = (v1 - v3);
      q13 = cross(v13, v1);
      ps13 = permuted_inner_produtct(v13, q13, threadRay->dir, ray);

      v01 = (v0 - v1);
      q01 = cross(v01, v0);
      ps01 = permuted_inner_produtct(v01, q01, threadRay->dir, ray);

      v21 = (v2 - v1);
      q21 = cross(v21, v1);
      ps21 = permuted_inner_produtct(v21, q21, threadRay->dir, ray);

      v02 = - v02;
      v03 = - v03;
      v32 = - v32;
      */
      //Test against faces 0, 2, 3
      if((ps13 <= 0 && ps32 <= 0 && ps21 <= 0)){
        float3 u = make_float3(ps13, ps32, ps21) / (ps13 + ps32 + ps21);
        if(constMemory.debug) threadRay->acccolor = make_float4(1, 0, 0, 1);
        round = tetAdj.x;
        point = u.x * v2 +  u.y * v1 + u.z * v3;
        point.w = 1.0f;
      }
      else{
        if((-ps13 <= 0 && -ps01 <= 0 && ps03 <= 0)){
          float3 u = make_float3(-ps13, -ps01, ps03) / (-ps13 -ps01 + ps03);
          if(constMemory.debug) threadRay->acccolor = make_float4(0, 0, 1, 1);
          round = tetAdj.z;
          point = u.x * v0 +  u.y * v3 + u.z * v1;
          point.w = 1.0f;
        }
        else{
          if((-ps21 <= 0 && -ps02 <= 0 && ps01 <= 0)){
            float3 u = make_float3(-ps21, -ps02, ps01) / (-ps21 -ps02 + ps01);
            if(constMemory.debug) threadRay->acccolor = make_float4(1, 0, 1, 1);
            round = tetAdj.w;
            point = u.x * v0 +  u.y * v1 + u.z * v2;
            point.w = 1.0f;
          }
        } 
      }
    }
    else if(threadRay->frontface == 2.0f){
      //Face 2: 3, 0, 1
      /*
      v2 = tex1Dfetch(texNode2, threadRay->frontid);

      v02 = (v0 - v2);
      q02 = cross(v02, v0);
      ps02 = permuted_inner_produtct(v02, q02, threadRay->dir, ray);

      v32 = (v3 - v2);
      q32 = cross(v32, v2);
      ps32 = permuted_inner_produtct(v32, q32, threadRay->dir, ray);

      v21 = (v2 - v1);
      q21 = cross(v21, v1);
      ps21 = permuted_inner_produtct(v21, q21, threadRay->dir, ray);

      v01 = - v01;
      v03 = - v03;
      v13 = - v13;
      */
      //Test against faces 0, 1, 3
      if((ps13 <= 0 && ps32 <= 0 && ps21 <= 0)){
        float3 u = make_float3(ps13, ps32, ps21) / (ps13 + ps32 + ps21);
        if(constMemory.debug) threadRay->acccolor = make_float4(1, 0, 0, 1);
        round = tetAdj.x;
        point = u.x * v2 +  u.y * v1 + u.z * v3;
        point.w = 1.0f;
      }
      else{
        if((-ps32 <= 0 && -ps03 <= 0 && ps02<= 0)){
          float3 u = make_float3(-ps32, -ps03, ps02) / (-ps32 -ps03 + ps02);
          if(constMemory.debug) threadRay->acccolor = make_float4(0, 1, 0, 1);
          round = tetAdj.y;
          point = u.x * v0 +  u.y * v2 + u.z * v3;
          point.w = 1.0f;
        }
        else{
          if((-ps21 <= 0 && -ps02 <= 0 && ps01 <= 0)){
            float3 u = make_float3(-ps21, -ps02, ps01) / (-ps21 -ps02 + ps01);
            if(constMemory.debug) threadRay->acccolor = make_float4(1, 0, 1, 1);
            round = tetAdj.w;
            point = u.x * v0 +  u.y * v1 + u.z * v2;
            point.w = 1.0f;
          }
        } 
      }
    }
    else if(threadRay->frontface == 3.0f){
      //Face 3: 1, 0, 2

      //threadRay->acccolor = make_float4(1, 0, 0, 1);
      //return;
      /*
      v3 = tex1Dfetch(texNode3, threadRay->frontid);

      v32 = (v3 - v2);
      q32 = cross(v32, v2);
      ps32 = permuted_inner_produtct(v32, q32, threadRay->dir, ray);

      v03 = (v0 - v3);
      q03 = cross(v03, v0);
      ps03 = permuted_inner_produtct(v03, q03, threadRay->dir, ray);

      v13 = (v1 - v3);
      q13 = cross(v13, v1);
      ps13 = permuted_inner_produtct(v13, q13, threadRay->dir, ray);

      v01 = - v01;
      v02 = - v02;
      v21 = - v21;
      */
      //Test against faces 0, 1, 2
      if((ps13 <= 0 && ps32 <= 0 && ps21 <= 0)){
        float3 u = make_float3(ps13, ps32, ps21) / (ps13 + ps32 + ps21);
        if(constMemory.debug) threadRay->acccolor = make_float4(1, 0, 0, 1);
        round = tetAdj.x;
        point = u.x * v2 +  u.y * v1 + u.z * v3;
        point.w = 1.0f;
      }
      else{
        if((-ps32 <= 0 && -ps03 <= 0 && ps02 <= 0)){
          float3 u = make_float3(-ps32, -ps03, ps02) / (-ps32 -ps03 + ps02);
          if(constMemory.debug) threadRay->acccolor = make_float4(0, 1, 0, 1);
          round = tetAdj.y;
          point = u.x * v0 +  u.y * v2 + u.z * v3;
          point.w = 1.0f;
        }
        else{
          if((-ps13 <= 0 && -ps01 <= 0 && ps03 <= 0)){
            float3 u = make_float3(-ps13, -ps01, ps03) / (-ps13 -ps01 + ps03);
            if(constMemory.debug) threadRay->acccolor = make_float4(0, 0, 1, 1);
            round = tetAdj.z;
            point = u.x * v0 +  u.y * v3 + u.z * v1;
            point.w = 1.0f;
          }
        } 
      }
    }


    t = length(point - threadRay->eyepos);
    //threadRay->acccolor = make_float4(t, t, t, 1);

    //threadRay->acccolor = make_float4(normalize(point - eyePos).x, normalize(point - eyePos).y, normalize(point - eyePos).z, 1);
    //return;

#else

    /*Face 0*/
    if(threadRay->frontface != 0){
      //Triangle 0
      planeEq = tex1Dfetch(texFace0Eq, threadRay->frontid);
      sameDirection = dot(threadRay->dir, planeEq);
      if(sameDirection > 0){
        sameDirection = - dot(planeEq, threadRay->eyepos) / sameDirection;
        if(sameDirection < t){
          t = sameDirection;
          if(constMemory.debug > 0) threadRay->acccolor = make_float4(1, 0, 0, 1);
          //threadRay->accColor = make_float4(0.5, 0.5, 0.5, 1);
          round = tetAdj.x;
        }
      }
    }

    /*Face 1*/
    if(threadRay->frontface != 1){
      //Triangle 0
      planeEq = tex1Dfetch(texFace1Eq, threadRay->frontid);
      sameDirection = dot(threadRay->dir, planeEq);
      if(sameDirection > 0){
        sameDirection = - dot(planeEq, threadRay->eyepos) / sameDirection;
        if(sameDirection < t){
          t = sameDirection;
          if(constMemory.debug > 0) threadRay->acccolor = make_float4(0, 1, 0, 1);
          //threadRay->accColor = make_float4(0.5, 0.5, 0.5, 1);
          round = tetAdj.y;
        }
      }
    }

    /*Face 2*/
    if(threadRay->frontface != 2){
      //Triangle 0
      planeEq = tex1Dfetch(texFace2Eq, threadRay->frontid);
      sameDirection = dot(threadRay->dir, planeEq);
      if(sameDirection > 0){
        sameDirection = - dot(planeEq, threadRay->eyepos) / sameDirection;
        if(sameDirection < t){
          t = sameDirection;
          if(constMemory.debug > 0) threadRay->acccolor = make_float4(0, 0, 1, 1);
          //threadRay->accColor = make_float4(0.5, 0.5, 0.5, 1);
          round = tetAdj.z;
        }
      }
    }

    /*Face 3*/
    if(threadRay->frontface != 3){
      //Triangle 0
      planeEq = tex1Dfetch(texFace3Eq, threadRay->frontid);
      sameDirection = dot(threadRay->dir, planeEq);
      if(sameDirection > 0){
        sameDirection = - dot(planeEq, threadRay->eyepos) / sameDirection;
        if(sameDirection < t){
          t = sameDirection;
          if(constMemory.debug > 0) threadRay->acccolor = make_float4(1, 1, 0, 1);
          //threadRay->accColor = make_float4(0.5, 0.5, 0.5, 1);
          round = tetAdj.w;
        }
      }
    }
    /*
    float4 tempdir;
    if(t >= 0 && t < HIP_INF_F){
    tempdir = (threadRay->eyepos + threadRay->dir * t) - threadRay->eyepos;
    }
    else{
    tempdir = make_float4(0, 0, 0, 0);
    }
    tempdir = normalize(tempdir);
    threadRay->acccolor = make_float4(tempdir.x, tempdir.y, tempdir.z, 1);
    return;
    */
#endif
#endif

    int rounded = floor(round + 0.5f);
#ifdef CUDARC_HEX
    backid = rounded / 6; 
    backfaceid = rounded % 6; 
#else
    backid = rounded / 4; 
    backfaceid = rounded % 4; 
#endif

    //return;


    //Round
    t = fmaxf(t, threadRay->t);


    float tetraBackScalar;
    tetraBackScalar = FindScalar(threadRay, t);

    if(constMemory.interpoltype == Step){
      float frontt = threadRay->t;
      float step = (t - frontt) / constMemory.numsteps;
      float frontscalar = threadRay->frontscalar;
      float backscalar = tetraBackScalar;
      for(int i=0; i<constMemory.numsteps; i++){
        IntegrateRayConst(threadRay, threadRay->eyepos, step / constMemory.maxedge, frontscalar, backscalar);
        frontt += step; 

        frontscalar = backscalar;
        backscalar = FindScalar(threadRay, frontt);

        if(threadRay->acccolor.w > 0.99)
          break;
      }
    }
    else{
      float isocpscalar = 3.0f;
      float volcpscalar = 3.0f;

      if(constMemory.isosurface)
        isocpscalar = FindControlPoint(threadRay, tetraBackScalar, tex1D(texIsoControlPoints, threadRay->frontscalar));

      if(constMemory.volumetric)
        volcpscalar = FindControlPoint(threadRay, tetraBackScalar, tex1D(texVolumetricControlPoints, threadRay->frontscalar));
      

      //Find if tetra contains cp
      float diffisocpfront = fabs(isocpscalar - threadRay->frontscalar);
      float diffvolcpfront = fabs(volcpscalar - threadRay->frontscalar);
      float diffbackfront = fabs(tetraBackScalar - threadRay->frontscalar);

      float diffcpfront = 3.0f;
      float cpscalar = 3.0f;
      if(diffisocpfront - diffvolcpfront < 1e-6){
        diffcpfront = diffisocpfront;
        cpscalar = isocpscalar;
      }
      else{
        diffcpfront = diffvolcpfront;
        cpscalar = volcpscalar;
      }

      if(diffcpfront - diffbackfront < 1e-6){
        //Integrate between front and iso, but dont traverse to the next tetra
        backid = threadRay->frontid;
        backfaceid = threadRay->frontface;
        tetraBackScalar = cpscalar;
        
        t = FindIntegrationStep(threadRay, t, diffcpfront, diffbackfront);
        
        /*
        float c0 = threadRay->currentelem.interpolfunc1.w;
        float c1 = threadRay->currentelem.interpolfunc0.x;
        float c2 = threadRay->currentelem.interpolfunc0.y;
        float c3 = threadRay->currentelem.interpolfunc0.z;
        float c4 = threadRay->currentelem.interpolfunc0.w;
        float c5 = threadRay->currentelem.interpolfunc1.x;
        float c6 = threadRay->currentelem.interpolfunc1.y;
        float c7 = threadRay->currentelem.interpolfunc1.z;

        float ox = threadRay->eyepos.x;
        float oy = threadRay->eyepos.y;
        float oz = threadRay->eyepos.z;

        float dx = threadRay->dir.x;
        float dy = threadRay->dir.y;
        float dz = threadRay->dir.z;

        float a = c0 + c1*ox + c2*oy + c4*ox*oy + c3*oz + c6*ox*oz + c5*oy*oz + c7*ox*oy*oz;
        float b = c1*dx + c2*dy + c3*dz + c4*dy*ox + c6*dz*ox + c4*dx*oy + c5*dz*oy + c7*dz*ox*oy + c6*dx*oz + c5*dy*oz + c7*dy*ox*oz + c7*dx*oy*oz;
        float c = c4*dx*dy + c6*dx*dz + c5*dy*dz + c7*dy*dz*ox + c7*dx*dz*oy + c7*dx*dy*oz;
        float d = c7*dx*dy*dz - cpscalar;


        float delta = 18.0f * a * b * c * d + 4.0f * b * b * b * d + b * b + c * c - 4 * a * c * c * c - 27.0f * a * a * d * d;

        if(delta > 0.0f)
          threadRay->acccolor = make_float4(1, 0, 0, 1);
        else if(delta == 0)
          threadRay->acccolor = make_float4(0, 1, 0, 1);
        else
          threadRay->acccolor = make_float4(0, 0, 1, 1);
        return;
        */
      }
      

      //Volumetric
      if(constMemory.volumetric > 0){
        float diff = t - threadRay->t;
        if(constMemory.debug == 0 && diff > 0){

          if(constMemory.interpoltype == Const){
            IntegrateRayConst(threadRay, threadRay->eyepos, diff / constMemory.maxedge, tetraBackScalar, threadRay->frontscalar);
          }
          else{
            IntegrateRayLinear(threadRay, threadRay->eyepos, diff / constMemory.maxedge, tetraBackScalar, threadRay->frontscalar);
          }
        }
      }
      
      //Isosurface
      if(constMemory.isosurface > 0 && backid == threadRay->frontid && cpscalar == isocpscalar){

#ifdef CUDARC_GRADIENT_PERVERTEX
        //Initialize barycentric interpolation
        float4 gradv0   = tex1Dfetch(texGrad0, threadRay->frontid);
        float4 gradv1   = tex1Dfetch(texGrad1, threadRay->frontid);
        float4 gradv2   = tex1Dfetch(texGrad2, threadRay->frontid);
        float4 gradv3   = tex1Dfetch(texGrad3, threadRay->frontid);
        float4 gradient = (gradv0 + gradv1 + gradv2 + gradv3)/4.0;
        float3 N = normalize(make_float3(gradient.x,gradient.y,gradient.z));
#else
        float3 N = normalize(make_float3(threadRay->currentelem.interpolfunc0));
#endif

        float4 color = tex1D(texIsoColorScale, tetraBackScalar);
        float3 L = normalize(make_float3(- threadRay->t * threadRay->dir));
        color.x *= abs(dot(N, L));
        color.y *= abs(dot(N, L));
        color.z *= abs(dot(N, L));
        color.x *= color.w; color.y *= color.w; color.z *= color.w;
        
        threadRay->acccolor += (1.0f - threadRay->acccolor.w) * color;
        //threadRay->acccolor = make_float4(make_float3(color), 1);
        //threadRay->acccolor = make_float4(1, 0, 0, 1);
      }
    }

    //threadRay->acccolor.w = 1;

    //Traverse
    threadRay->frontid = backid;
    threadRay->frontface = backfaceid;
    threadRay->frontscalar = tetraBackScalar;
    threadRay->t = t;

    threadRay->currentelem.interpolfunc0 = tex1Dfetch(texInterpolFunc0, threadRay->frontid);
#ifdef CUDARC_HEX
    threadRay->currentelem.interpolfunc1 = tex1Dfetch(texInterpolFunc1, threadRay->frontid);
    hexAdj1 = tex1Dfetch(texAdj0, threadRay->frontid);
    hexAdj2 = tex1Dfetch(texAdj1, threadRay->frontid);
#else
    tetAdj = tex1Dfetch(texAdj0, threadRay->frontid);
#endif
    t = HIP_INF_F;
    backid = 0;
    backfaceid = 0;
    round = 0;
  }
}

/**
* Init CUDA variables
*/
extern "C" void init(GLuint p_handleTexIntersect, GLuint p_handlePboOutput){




  //Prop
  hipDeviceProp_t prop;
  int dev;
  memset(&prop, 0, sizeof(hipDeviceProp_t));
  prop.major = 1;
  prop.minor = 1;
  CUDA_SAFE_CALL(hipChooseDevice(&dev, &prop));
  CUDA_SAFE_CALL(cudaGLSetGLDevice( dev ));


  //Register output buffer
  CUDA_SAFE_CALL(hipGraphicsGLRegisterBuffer(&cudaPboHandleOutput, p_handlePboOutput, cudaGraphicsMapFlagsNone));


  //Debug
  //CUDA_SAFE_CALL(hipMalloc((void**)&dev_debug, sizex * sizey * sizeof(float4)));

}

/**
* CUDA callback (device)
*/
__global__ void Run(int depthPeelPass, float4 eyePos, float4* dev_outputData){

  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  int offset = x + y * blockDim.x * gridDim.x;

  Ray threadRay = Initialize(x, y, offset, eyePos);

  if(depthPeelPass > 0)
    threadRay.acccolor = dev_outputData[offset];
  

  if(threadRay.frontid != 0)
    Traverse(x, y, offset, &threadRay);


  dev_outputData[offset] = threadRay.acccolor;
}

/**
* CUDA callback (host)
*/
extern "C" void run(float* p_kernelTime, float* p_overheadTime, int depthPeelPass, float* p_eyePos, int handleTexIntersect){

#ifdef CUDARC_TIME
  hipEvent_t start, stop;
  CUDA_SAFE_CALL(hipEventCreate(&start));
  CUDA_SAFE_CALL(hipEventCreate(&stop));
  CUDA_SAFE_CALL(hipEventRecord(start, 0));
#endif

  //Register intersect buffer
  //TODO: do it every frame? Possible over-head?
  CUDA_SAFE_CALL(hipGraphicsGLRegisterImage(&cudaTexHandleIntersect, handleTexIntersect, GL_TEXTURE_2D, cudaGraphicsMapFlagsReadOnly));

  size_t size;
  CUDA_SAFE_CALL(hipGraphicsMapResources(1, &cudaPboHandleOutput, NULL));
  CUDA_SAFE_CALL(hipGraphicsResourceGetMappedPointer((void**)&dev_outputData, &size, cudaPboHandleOutput));

  CUDA_SAFE_CALL(hipGraphicsMapResources(1, &cudaTexHandleIntersect, NULL));
  CUDA_SAFE_CALL(hipGraphicsSubResourceGetMappedArray(&dev_intersectData, cudaTexHandleIntersect, 0, 0));


  //TODO: replace the rounding of the texIntersect values with the following code
  texIntersect.addressMode[0] = hipAddressModeClamp;
  texIntersect.addressMode[1] = hipAddressModeClamp;
  texIntersect.filterMode = hipFilterModePoint;
  texIntersect.normalized = false;
  CUDA_SAFE_CALL(hipBindTextureToArray(texIntersect, dev_intersectData));

  texVolumetricColorScale.addressMode[0] = hipAddressModeClamp;
  texVolumetricColorScale.filterMode = hipFilterModeLinear;
  texVolumetricColorScale.normalized = true;
  CUDA_SAFE_CALL(hipBindTextureToArray(texVolumetricColorScale, dev_volcolorscale));

  texIsoColorScale.addressMode[0] = hipAddressModeClamp;
  texIsoColorScale.filterMode = hipFilterModeLinear;
  texIsoColorScale.normalized = true;
  CUDA_SAFE_CALL(hipBindTextureToArray(texIsoColorScale, dev_isocolorscale));

  texVolumetricControlPoints.addressMode[0] = hipAddressModeClamp;
  texVolumetricControlPoints.filterMode = hipFilterModeLinear;
  texVolumetricControlPoints.normalized = true;
  CUDA_SAFE_CALL(hipBindTextureToArray(texVolumetricControlPoints, dev_volcontrolpoints));

  texIsoControlPoints.addressMode[0] = hipAddressModeClamp;
  texIsoControlPoints.filterMode = hipFilterModeLinear;
  texIsoControlPoints.normalized = true;
  CUDA_SAFE_CALL(hipBindTextureToArray(texIsoControlPoints, dev_isocontrolpoints));


  texZetaPsiGamma.addressMode[0] = hipAddressModeClamp;
  texZetaPsiGamma.addressMode[1] = hipAddressModeClamp;
  texZetaPsiGamma.filterMode = hipFilterModeLinear;
  texZetaPsiGamma.normalized = true;
  CUDA_SAFE_CALL(hipBindTextureToArray(texZetaPsiGamma, dev_zetaPsiGamma));


  //Camera settings
  float4 eyePos = make_float4(p_eyePos[0], p_eyePos[1], p_eyePos[2], 1);

#ifdef CUDARC_TIME
  CUDA_SAFE_CALL(hipEventRecord(stop, 0));
  CUDA_SAFE_CALL(hipEventSynchronize(stop));
  CUDA_SAFE_CALL(hipEventElapsedTime(p_overheadTime, start, stop));

  CUDA_SAFE_CALL(hipEventRecord(start, 0));
#endif

  //Kernel call
  Run<<<grids, threads>>>(depthPeelPass, eyePos, dev_outputData);


  CUDA_SAFE_CALL(hipGraphicsUnmapResources( 1, &cudaPboHandleOutput, NULL ) );
  CUDA_SAFE_CALL(hipGraphicsUnmapResources( 1, &cudaTexHandleIntersect, NULL ) );
  CUDA_SAFE_CALL(hipGraphicsUnregisterResource(cudaTexHandleIntersect));


#ifdef CUDARC_TIME
  CUDA_SAFE_CALL(hipEventRecord(stop, 0));
  CUDA_SAFE_CALL(hipEventSynchronize(stop));
  CUDA_SAFE_CALL(hipEventElapsedTime(p_kernelTime, start, stop));

  CUDA_SAFE_CALL(hipEventDestroy(start));
  CUDA_SAFE_CALL(hipEventDestroy(stop));
#endif

#ifdef CUDARC_VERBOSE
  hipError_t cudaLastError = hipGetLastError();
  if(cudaLastError != hipSuccess)
    printf("%s\n", hipGetErrorString(cudaLastError));
#endif

}

/**
* Create adj textures on the GPU
*/
extern "C" void createGPUAdjTex(int index, int size, float* data){
  CUDA_SAFE_CALL(hipMalloc((void**)&dev_adj[index], size));
  CUDA_SAFE_CALL(hipMemcpy(dev_adj[index], data, size, hipMemcpyHostToDevice));

  switch(index)
  {
  case 0:
    CUDA_SAFE_CALL(hipBindTexture(0, texAdj0, dev_adj[index], size));
    break;
#ifdef CUDARC_HEX
  case 1:
    CUDA_SAFE_CALL(hipBindTexture(0, texAdj1, dev_adj[index], size));
    break;
#endif
  default:
    break;
  }

#ifdef CUDARC_VERBOSE
  printf("Adj. to CUDA: %s\n", hipGetErrorString(hipGetLastError()));
#endif
}


/**
* Create gradient vertex textures on the GPU
*/
#ifdef CUDARC_GRADIENT_PERVERTEX
extern "C" void createGPUGradientVertexTex(int ni, int size, float* data)
{
  CUDA_SAFE_CALL(hipMalloc((void**)&dev_gradientVertex[ni], size));
  CUDA_SAFE_CALL(hipMemcpy(dev_gradientVertex[ni], data, size, hipMemcpyHostToDevice));

  switch(ni)
  {
  case 0:
    CUDA_SAFE_CALL(hipBindTexture(0, texGrad0, dev_gradientVertex[ni], size));
    break;
  case 1:
    CUDA_SAFE_CALL(hipBindTexture(0, texGrad1, dev_gradientVertex[ni], size));
    break;
  case 2:
    CUDA_SAFE_CALL(hipBindTexture(0, texGrad2, dev_gradientVertex[ni], size));
    break;
  case 3:
    CUDA_SAFE_CALL(hipBindTexture(0, texGrad3, dev_gradientVertex[ni], size));
    break;
  default:
    break;
  }

#ifdef CUDARC_VERBOSE
  printf("Gradient vertex to CUDA: %s\n", hipGetErrorString(hipGetLastError()));
#endif

}
#endif

/**
* Create node textures on the GPU
*/
#if defined(CUDARC_PLUCKER) || defined(CUDARC_BILINEAR)
extern "C" void createGPUCollisionTex(int ni, int size, float* data){
  CUDA_SAFE_CALL(hipMalloc((void**)&dev_collision[ni], size));
  CUDA_SAFE_CALL(hipMemcpy(dev_collision[ni], data, size, hipMemcpyHostToDevice));

  switch(ni)
  {
  case 0:
    CUDA_SAFE_CALL(hipBindTexture(0, texNode0, dev_collision[ni], size));
    break;
  case 1:
    CUDA_SAFE_CALL(hipBindTexture(0, texNode1, dev_collision[ni], size));
    break;
  case 2:
    CUDA_SAFE_CALL(hipBindTexture(0, texNode2, dev_collision[ni], size));
    break;
  case 3:
    CUDA_SAFE_CALL(hipBindTexture(0, texNode3, dev_collision[ni], size));
    break;
#ifdef CUDARC_HEX
  case 4:
    CUDA_SAFE_CALL(hipBindTexture(0, texNode4, dev_collision[ni], size));
    break;
  case 5:
    CUDA_SAFE_CALL(hipBindTexture(0, texNode5, dev_collision[ni], size));
    break;
  case 6:
    CUDA_SAFE_CALL(hipBindTexture(0, texNode6, dev_collision[ni], size));
    break;
  case 7:
    CUDA_SAFE_CALL(hipBindTexture(0, texNode7, dev_collision[ni], size));
    break;
#endif
  default:
    break;
  }

#ifdef CUDARC_VERBOSE
  printf("Collision data to CUDA: %s\n", hipGetErrorString(hipGetLastError()));
#endif

}
#else
/**
* Create face textures on the GPU
*/
extern "C" void createGPUCollisionTex(int fi, int size, float* data){
  CUDA_SAFE_CALL(hipMalloc((void**)&dev_collision[fi], size));
  CUDA_SAFE_CALL(hipMemcpy(dev_collision[fi], data, size, hipMemcpyHostToDevice));

  switch(fi)
  {
  case 0:
    CUDA_SAFE_CALL(hipBindTexture(0, texFace0Eq, dev_collision[fi], size));
    break;
  case 1:
    CUDA_SAFE_CALL(hipBindTexture(0, texFace1Eq, dev_collision[fi], size));
    break;
  case 2:
    CUDA_SAFE_CALL(hipBindTexture(0, texFace2Eq, dev_collision[fi], size));
    break;
  case 3:
    CUDA_SAFE_CALL(hipBindTexture(0, texFace3Eq, dev_collision[fi], size));
    break;
#ifdef CUDARC_HEX
  case 4:
    CUDA_SAFE_CALL(hipBindTexture(0, texFace4Eq, dev_collision[fi], size));
    break;
  case 5:
    CUDA_SAFE_CALL(hipBindTexture(0, texFace5Eq, dev_collision[fi], size));
    break;
#endif
  default:
    break;
  }

#ifdef CUDARC_VERBOSE
  printf("Collision data to CUDA: %s\n", hipGetErrorString(hipGetLastError()));
#endif

}
#endif

/**
* Create interpolation function textures on the GPU
*/
extern "C" void createGPUInterpolFuncTex(int index, int size, float* data){
  CUDA_SAFE_CALL(hipMalloc((void**)&dev_interpolfunc[index], size));
  CUDA_SAFE_CALL(hipMemcpy(dev_interpolfunc[index], data, size, hipMemcpyHostToDevice));

  switch(index)
  {
  case 0:
    CUDA_SAFE_CALL(hipBindTexture(0, texInterpolFunc0, dev_interpolfunc[index], size));
    break;
#ifdef CUDARC_HEX
  case 1:
    CUDA_SAFE_CALL(hipBindTexture(0, texInterpolFunc1, dev_interpolfunc[index], size));
    break;
#endif
  default:
    break;
  }

#ifdef CUDARC_VERBOSE
  printf("Interpolation func. to CUDA: %s\n", hipGetErrorString(hipGetLastError()));
#endif
}

/**
* Create color scale texture on the GPU
*/
extern "C" void createGPUColorScaleTex(int numValues, int size, float* volcolorscaledata, float* isocolorscale){
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();

  CUDA_SAFE_CALL(hipMallocArray(&dev_volcolorscale, &channelDesc, numValues));
  CUDA_SAFE_CALL(hipMemcpyToArray(dev_volcolorscale, 0, 0, volcolorscaledata, size, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipBindTextureToArray(texVolumetricColorScale, dev_volcolorscale, channelDesc));  

  CUDA_SAFE_CALL(hipMallocArray(&dev_isocolorscale, &channelDesc, numValues));
  CUDA_SAFE_CALL(hipMemcpyToArray(dev_isocolorscale, 0, 0, isocolorscale, size, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipBindTextureToArray(texIsoColorScale, dev_isocolorscale, channelDesc));  

#ifdef CUDARC_VERBOSE
  printf("Color scales to CUDA: %s\n", hipGetErrorString(hipGetLastError()));
#endif
}

/**
* Create control points texture on the GPU
*/
extern "C" void createGPUVolControlPointsTex(int numValues, int size, float* data){
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
  CUDA_SAFE_CALL(hipMallocArray(&dev_volcontrolpoints, &channelDesc, numValues));
  CUDA_SAFE_CALL(hipMemcpyToArray(dev_volcontrolpoints, 0, 0, data, size, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipBindTextureToArray(texVolumetricColorScale, dev_volcontrolpoints, channelDesc)); 

#ifdef CUDARC_VERBOSE
  printf("Control points to CUDA: %s\n", hipGetErrorString(hipGetLastError()));
#endif
}

extern "C" void createGPUIsoControlPointsTex(int numValues, int size, float* data){
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
  CUDA_SAFE_CALL(hipMallocArray(&dev_isocontrolpoints, &channelDesc, numValues));
  CUDA_SAFE_CALL(hipMemcpyToArray(dev_isocontrolpoints, 0, 0, data, size, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipBindTextureToArray(texIsoColorScale, dev_isocontrolpoints, channelDesc)); 

#ifdef CUDARC_VERBOSE
  printf("Control points to CUDA: %s\n", hipGetErrorString(hipGetLastError()));
#endif
}

/**
* Create zetapsigamma texture on the GPU
*/
extern "C" void createGPUZetaPsiGammaTex(int numValues, int size, float* data){
#ifdef CUDARC_HEX
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
#else
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
#endif

  CUDA_SAFE_CALL(hipMallocArray(&dev_zetaPsiGamma, &channelDesc, numValues, numValues));
  CUDA_SAFE_CALL(hipMemcpyToArray(dev_zetaPsiGamma, 0, 0, data, size, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipBindTextureToArray(texZetaPsiGamma, dev_zetaPsiGamma, channelDesc)); 

#ifdef CUDARC_VERBOSE
  printf("PsiGamma to CUDA: %s\n", hipGetErrorString(hipGetLastError()));
#endif
}

/**
* Delete textures from the GPU
*/
extern "C" void deleteGPUTextures(int numAdjTex, int numInterpolFuncTex){

  for(int i=0; i<numAdjTex; i++){
    if(i==0){
      CUDA_SAFE_CALL(hipUnbindTexture(texAdj0));
    }
#ifdef CUDARC_HEX
    else if(i==1){
      CUDA_SAFE_CALL(hipUnbindTexture(texAdj1));
    }
#endif
    CUDA_SAFE_CALL(hipFree(dev_collision[i]));
  }

  for(int i=0; i<numInterpolFuncTex; i++){

    if(i==0){
      CUDA_SAFE_CALL(hipUnbindTexture(texInterpolFunc0));
    }
#ifdef CUDARC_HEX
    else if(i==1){
      CUDA_SAFE_CALL(hipUnbindTexture(texInterpolFunc1));
    }
#endif
    CUDA_SAFE_CALL(hipFree(dev_collision[i]));
  }


  //Color scale
  CUDA_SAFE_CALL(hipUnbindTexture(texVolumetricColorScale));
  CUDA_SAFE_CALL(hipFree(dev_volcolorscale));

  //Iso surfaces
  CUDA_SAFE_CALL(hipUnbindTexture(texVolumetricControlPoints));
  CUDA_SAFE_CALL(hipFree(dev_volcontrolpoints));

  //Psi Gamma table
  CUDA_SAFE_CALL(hipUnbindTexture(texZetaPsiGamma));
  CUDA_SAFE_CALL(hipFree(dev_zetaPsiGamma));


}

/**
* Print memory usage
*/
extern "C" void printInfoGPUMemory(){
  unsigned int free, total;
  hipMemGetInfo(&free, &total);
  printf("#GPU Mem Info: Free = %d (%f), Total = %d\n", free, (float)free/(float)total, total);
}


/**
* Set const memory values
*/
extern "C" void update(int p_blocksizex, int p_blocksizey, int p_winsizex, int p_winsizey, bool p_debug, float p_maxedge, int p_interpoltype, int p_numsteps, int p_numtraverses, int p_numelem, bool p_isosurface, bool p_volumetric){

  grids = dim3(p_winsizex / p_blocksizex, p_winsizey / p_blocksizey);
  threads = dim3(p_blocksizex, p_blocksizey);

  ConstMemory *tempConstMemory = (ConstMemory*)malloc( sizeof(ConstMemory));
  tempConstMemory->numTets = p_numelem;
  tempConstMemory->screenSize = make_float2(p_winsizex, p_winsizey);
  tempConstMemory->maxedge = p_maxedge;
  tempConstMemory->interpoltype = p_interpoltype;
  tempConstMemory->numsteps = p_numsteps;
  tempConstMemory->numtraverses = p_numtraverses;
  tempConstMemory->debug = p_debug;
  tempConstMemory->isosurface = p_isosurface;
  tempConstMemory->volumetric = p_volumetric;

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constMemory), tempConstMemory, sizeof(ConstMemory)));
  delete tempConstMemory;
}
